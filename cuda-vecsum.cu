#include "hip/hip_runtime.h"
#include "book.h"
#include <cstdlib>
#include "cuda-vecsum.h"

/* Compute vector sum c=a+scale*b on GPU. a,b,c reside on GPU memory.
 * This function must be launched as a CUDA kernel */
__global__ void scalarVecSum( int* a, int* b, int N, int scale, int* c) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N){
        c[tid] = a[tid] + scale*b[tid];
		}
}

/* Compute vector sum c=a+scale*b using GPU. a,b,c reside in host memory, 
 * so this function can be called by normal C++ code*/
void addVectors(int* a, int* b, int N, int scale, int* c){
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

		// copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    scalarVecSum<<<N,1>>>( dev_a, dev_b, N, scale, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );


    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
    return;
}

