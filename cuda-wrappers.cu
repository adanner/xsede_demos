#include "hip/hip_runtime.h"
#include "book.h"
#include <cstdlib>
#include "cuda-wrappers.h"

/* Compute vector sum c=a+scale*b on GPU. a,b,c reside on GPU memory.
 * This function must be launched as a CUDA kernel */
__global__ void scalarVecSum( int* a, int* b, int N, int scale, int* c);

/* Return number of CUDA devices available on current host, 
 * returns 0 if error, or no devices found.*/
int getGPUCount(){
	int ans=0;
	if(hipGetDeviceCount(&ans) != hipSuccess){
		return 0;
	}
	return ans;
}

/* Select and set the current GPU for current thread/process id
 * Assumes the number of processes/threads per host is no greater
 * than the number of GPUs. Returns -1 if there is an error, and a 
 * valid GPU id if no error. The current GPU device will be set to the
 * valid GPU id, so the user does not need to set this after calling the
 * function */
int pickGPU(int rank, int ngpus){
	int gpuID = rank%ngpus; //any better way?
	if(hipSetDevice(gpuID) != hipSuccess){
		return -1;
	}
	return gpuID;
}

/* Compute vector sum c=a+scale*b using GPU. a,b,c reside in host memory, 
 * so this function can be called by normal C++ code*/
void addVectors(int* a, int* b, int N, int scale, int* c){
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

		// copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    scalarVecSum<<<N,1>>>( dev_a, dev_b, N, scale, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );


    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );
    return;
}

/* Compute vector sum c=a+scale*b on GPU. a,b,c reside on GPU memory.
 * This function must be launched as a CUDA kernel */
__global__ void scalarVecSum( int* a, int* b, int N, int scale, int* c) {
	// this thread handles the data at its thread id
	int tid = blockIdx.x;    
	if (tid < N){
		c[tid] = a[tid] + scale*b[tid];
	}
}


